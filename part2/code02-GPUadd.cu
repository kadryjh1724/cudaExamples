#include <stdio.h>
#include "hip/hip_runtime.h"


__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main(void) {
    int a, b, c;
    int *d_c;
    printf("Input two integers: ");
    scanf("%d %d", &a, &b);
    // Assign device memory
    hipMalloc((void**)&d_c, sizeof(int));
    // Kernel call
    add<<<1,1>>>(a, b, d_c);
    // Copy the result
    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d + %d = %d\n", a, b, c);
    // Free device memory
    hipFree(d_c);
    return 0;
}