#include <stdio.h>

#include "hip/hip_runtime.h"


__global__ void helloCUDA(void) {
    printf("Hello world from GPU, thread %d!\n", threadIdx.x);
}

int main(void) {
    printf("Hello world from CPU!\n");
    helloCUDA<<<1, 16>>>();
    hipDeviceSynchronize();
    return 0;
}