#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

int main(void) {
    hipDeviceProp_t prop;

    // Get total device number
    int count;
    hipGetDeviceCount(&count);

    for (int i = 0; i < count; i++) {
        hipGetDeviceProperties(&prop, i);
        printf("===== ===== ===== ===== ===== ===== ===== ===== =====\n");
        printf("----- [Device %02d]       %s\n", i, prop.name);
        printf("Compute capability      : %d.%d\n", prop.major, prop.minor);
        printf("Clock rate              : %d\n", prop.clockRate);
        printf("----- [Device %02d]       Memory information\n", i);
        printf("Total global memory     : %ld MB\n", prop.totalGlobalMem / (1024 * 1024));
        printf("Total const. memory     : %ld KB\n", prop.totalConstMem / 1024);
        printf("L2 Cache size           : %ld KB\n", prop.l2CacheSize / 1024);
        printf("----- [Device %02d]       Multiprocessor information\n", i);
        printf("Multiprocessor count    : %d\n", prop.multiProcessorCount);
        printf("Shared memory per block : %ld KB\n", prop.sharedMemPerBlock / 1024);
        printf("Shared memory per mp    : %ld KB\n", prop.sharedMemPerMultiprocessor / 1024);
        printf("Registers per mp        : %d\n", prop.regsPerBlock);
        printf("Num. of threads in warp : %d\n", prop.warpSize);
        printf("Max threads per block   : %d\n", prop.maxThreadsPerBlock);
        printf("Max thread dim.         : (%ld, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dim.           : (%ld, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("===== ===== ===== ===== ===== ===== ===== ===== =====\n\n");
    }
    return 0;
}